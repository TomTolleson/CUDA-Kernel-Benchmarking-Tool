#include <hip/hip_runtime.h>
#include <stdio.h>
#include "kernel.cuh"
#include "timer.hpp"

class CudaBenchmark {
private:
    int N;
    float *d_A, *d_B, *d_C;
    float *h_A, *h_B, *h_C;

public:
    CudaBenchmark(int size) : N(size) {
        // Allocate host memory
        h_A = new float[N];
        h_B = new float[N];
        h_C = new float[N];

        // Initialize data
        for(int i = 0; i < N; i++) {
            h_A[i] = rand() / (float)RAND_MAX;
            h_B[i] = rand() / (float)RAND_MAX;
        }

        // Allocate device memory
        hipMalloc(&d_A, N * sizeof(float));
        hipMalloc(&d_B, N * sizeof(float));
        hipMalloc(&d_C, N * sizeof(float));
    }

    void runBenchmark() {
        Timer timer;
        
        // Copy inputs to device
        hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice);

        // Launch kernel
        int threadsPerBlock = 256;
        int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
        
        timer.start();
        vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
        timer.stop();

        // Copy result back to host
        hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);
        
        printf("Kernel execution time: %f ms\n", timer.elapsedMilliseconds());
    }

    ~CudaBenchmark() {
        // Cleanup
        delete[] h_A;
        delete[] h_B;
        delete[] h_C;
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }
}; 