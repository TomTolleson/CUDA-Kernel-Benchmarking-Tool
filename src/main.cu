#include <hip/hip_runtime.h>
#include <stdio.h>
#include "benchmark.cu"

int main() {
    // Print device properties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Device: %s\n", prop.name);
    printf("Compute capability: %d.%d\n", prop.major, prop.minor);
    
    // Run benchmarks with different sizes
    int sizes[] = {1000000, 10000000, 100000000};
    
    for (int size : sizes) {
        printf("\nRunning benchmark with size %d\n", size);
        CudaBenchmark benchmark(size);
        benchmark.runBenchmark();
    }

    return 0;
} 