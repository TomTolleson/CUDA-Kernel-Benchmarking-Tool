#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "kernel.cuh"

__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

__global__ void computeKernel(float *d_data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float x = d_data[idx];
        for(int i = 0; i < 100; i++) {
            x = x * x + 1.0f;
        }
        d_data[idx] = x;
    }
}

__global__ void memoryKernel(float *d_in, float *d_out, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        d_out[idx] = d_in[idx];
    }
} 